// kernel: 4.36429 ms
// kernel: 17.2532 ms
// kernel: 47.4389 ms


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <zlib.h>
#include <png.h>

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE 8
#define BLOCK_X 32
#define BLOCK_Y 32
#define R_X (MASK_X / 2)
#define R_Y (MASK_Y / 2)
#define TILE_X (BLOCK_X + 2 * R_X)
#define TILE_Y (BLOCK_Y + 2 * R_Y)

/* Hint 7 */

int mask[MASK_N][MASK_X][MASK_Y] = {
  { { -1, -4, -6, -4, -1},
    { -2, -8, -12, -8, -2},
    {  0,  0,  0,  0,  0},
    {  2,  8, 12,  8,  2},
    {  1,  4,  6,  4,  1}
  },
  { { -1, -2,  0,  2,  1},
    { -4, -8,  0,  8,  4},
    { -6, -12,  0, 12,  6},
    { -4, -8,  0,  8,  4},
    { -1, -2,  0,  2,  1}
  }
};

int read_png(const char* filename, unsigned char** image, unsigned* height,
             unsigned* width, unsigned* channels) {

  unsigned char sig[8];
  FILE* infile;
  infile = fopen(filename, "rb");

  fread(sig, 1, 8, infile);
  if (!png_check_sig(sig, 8))
    return 1;   /* bad signature */

  png_structp png_ptr;
  png_infop info_ptr;

  png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
  if (!png_ptr)
    return 4;   /* out of memory */

  info_ptr = png_create_info_struct(png_ptr);
  if (!info_ptr) {
    png_destroy_read_struct(&png_ptr, NULL, NULL);
    return 4;   /* out of memory */
  }

  png_init_io(png_ptr, infile);
  png_set_sig_bytes(png_ptr, 8);
  png_read_info(png_ptr, info_ptr);
  int bit_depth, color_type;
  png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

  png_uint_32  i, rowbytes;
  png_bytep  row_pointers[*height];
  png_read_update_info(png_ptr, info_ptr);
  rowbytes = png_get_rowbytes(png_ptr, info_ptr);
  *channels = (int) png_get_channels(png_ptr, info_ptr);

  if ((*image = (unsigned char *) malloc(rowbytes * *height)) == NULL) {
    png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
    return 3;
  }

  for (i = 0;  i < *height;  ++i)
    row_pointers[i] = *image + i * rowbytes;
  png_read_image(png_ptr, row_pointers);
  png_read_end(png_ptr, NULL);
  return 0;
}

void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width,
               const unsigned channels) {
  FILE* fp = fopen(filename, "wb");
  png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
  png_infop info_ptr = png_create_info_struct(png_ptr);
  png_init_io(png_ptr, fp);
  png_set_IHDR(png_ptr, info_ptr, width, height, 8,
               PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
               PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
  png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
  png_write_info(png_ptr, info_ptr);
  png_set_compression_level(png_ptr, 1);

  png_bytep row_ptr[height];
  for (int i = 0; i < height; ++ i) {
    row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
  }
  png_write_image(png_ptr, row_ptr);
  png_write_end(png_ptr, NULL);
  png_destroy_write_struct(&png_ptr, &info_ptr);
  fclose(fp);
}

__global__ void sobel(
  const unsigned char* s,      // 來源影像
  unsigned char*       t,      // 輸出影像
  unsigned             height,
  unsigned             width,
  unsigned             channels,
  const int*           d_mask) // filter mask
{
  // 1) 把 mask 複製到 shared memory（block 內一次做完）
  __shared__ int mask_s[MASK_N][MASK_X][MASK_Y];
  int tid_mask = threadIdx.y * BLOCK_X + threadIdx.x;
  int mask_elems = MASK_N * MASK_X * MASK_Y;
  int mask_stride = BLOCK_X * BLOCK_Y;
  for (int k = tid_mask; k < mask_elems; k += mask_stride) {
    int n   = k / (MASK_X * MASK_Y);
    int rem = k % (MASK_X * MASK_Y);
    int i   = rem / MASK_Y;
    int j   = rem % MASK_Y;
    mask_s[n][i][j] = d_mask[k];
  }
  __syncthreads();

  // 2) 把 tile (包含 halo) 載入 shared memory
  __shared__ unsigned char tile[TILE_Y][TILE_X][3];
  int tid_tile = threadIdx.y * BLOCK_X + threadIdx.x;
  int tile_elems = TILE_X * TILE_Y;
  int tile_stride = BLOCK_X * BLOCK_Y;
  for (int k = tid_tile; k < tile_elems; k += tile_stride) {
    int dy = k / TILE_X;
    int dx = k % TILE_X;
    int gx = blockIdx.x * BLOCK_X + dx - R_X;
    int gy = blockIdx.y * BLOCK_Y + dy - R_Y;

    unsigned char b = 0, g = 0, r = 0;
    if (gx >= 0 && gx < width && gy >= 0 && gy < height) {
      int idx = (gy * width + gx) * channels;
      b = s[idx + 0];
      g = s[idx + 1];
      r = s[idx + 2];
    }
    tile[dy][dx][0] = b;
    tile[dy][dx][1] = g;
    tile[dy][dx][2] = r;
  }
  __syncthreads();

  // 3) 每個 thread 處理自己那個 output pixel
  int x = blockIdx.x * BLOCK_X + threadIdx.x;
  int y = blockIdx.y * BLOCK_Y + threadIdx.y;
  if (x >= width || y >= height) return;

  // Sobel 卷積
  float val[MASK_N * 3] = {0.0f};
  int adjX = MASK_X % 2, adjY = MASK_Y % 2;
  for (int m = 0; m < MASK_N; ++m) {
    for (int v = -R_Y; v < R_Y + adjY; ++v) {
      for (int u = -R_X; u < R_X + adjX; ++u) {
        int lx = threadIdx.x + R_X + u;
        int ly = threadIdx.y + R_Y + v;
        int w  = mask_s[m][u + R_X][v + R_Y];

        unsigned char B = tile[ly][lx][0];
        unsigned char G = tile[ly][lx][1];
        unsigned char R = tile[ly][lx][2];

        val[m * 3 + 0] += B * w;
        val[m * 3 + 1] += G * w;
        val[m * 3 + 2] += R * w;
      }
    }
  }

  // 4) 合併、clamp、寫回
  float sumB = 0, sumG = 0, sumR = 0;
  for (int m = 0; m < MASK_N; ++m) {
    sumB += val[m * 3 + 0] * val[m * 3 + 0];
    sumG += val[m * 3 + 1] * val[m * 3 + 1];
    sumR += val[m * 3 + 2] * val[m * 3 + 2];
  }
  sumB = sqrt(sumB) / SCALE; sumG = sqrt(sumG) / SCALE; sumR = sqrt(sumR) / SCALE;
  unsigned char cB = sumB > 255 ? 255 : (unsigned char)sumB;
  unsigned char cG = sumG > 255 ? 255 : (unsigned char)sumG;
  unsigned char cR = sumR > 255 ? 255 : (unsigned char)sumR;

  int outIdx = (y * width + x) * channels;
  t[outIdx + 0] = cB;
  t[outIdx + 1] = cG;
  t[outIdx + 2] = cR;
}

int main(int argc, char** argv) {
  assert(argc == 3);
  unsigned height, width, channels;
  unsigned char* host_s = NULL;
  read_png(argv[1], &host_s, &height, &width, &channels);
  unsigned char* host_t = (unsigned char*) malloc(height * width * channels * sizeof(unsigned char));

  unsigned char *device_s, *device_t;
  int *d_mask;
  hipMalloc((void**)&device_s, height * width * channels * sizeof(unsigned char));
  hipMalloc((void**)&device_t, height * width * channels * sizeof(unsigned char));
  hipMalloc((void**)&d_mask, MASK_N * MASK_X * MASK_Y * sizeof(int));

  hipMemcpy(device_s, host_s, height * width * channels * sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMemcpy(d_mask, mask, MASK_N * MASK_X * MASK_Y * sizeof(int), hipMemcpyHostToDevice);

  dim3 block(BLOCK_X, BLOCK_Y);
  dim3 grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y);
  sobel<<<grid, block>>>(device_s, device_t, height, width, channels, d_mask);
  hipDeviceSynchronize();

  hipMemcpy(host_t, device_t, height * width * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);
  write_png(argv[2], host_t, height, width, channels);

  // Clean up
  hipFree(device_s);
  hipFree(device_t);
  free(host_s);
  free(host_t);

  return 0;
}
